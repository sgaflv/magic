
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>       /* time */
#include <iostream>
#include <chrono>
#include <unistd.h>


typedef unsigned long long int u64;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                 
typedef unsigned long long int BBoard;

#define INDEX_BITS 11
#define IDX 61
#define INITIAL_COUNTER 0x0007ffc000000000ULL

//#define INDEX_BITS 10
//#define IDX 62
//#define INITIAL_COUNTER 0x0003ffef00000000ULL

//#define INDEX_BITS 10
//#define IDX 55
//#define INITIAL_COUNTER 0x510FFFF5F0000000ULL


#define ELEM_COUNT (1ULL << INDEX_BITS)

__device__ void add_bit(BBoard &board, int x, int y) {
    board |= 1ull << (x + y * 8);
}

__device__ bool has_bit(BBoard &board, int x, int y) {
    return (board & (1ull << (x + y * 8))) > 0;
}

__device__ bool d_has_bit(BBoard &board, int x, int y) {
    return (board & (1ull << (x + y * 8))) > 0;
}

__device__ void bb_print(BBoard board) {

        printf("\n");

    for (int y = 0; y < 8; y++) {
        for (int x = 0; x < 8; x++) {
            if (d_has_bit(board, x, 7-y)) {
                printf("*");
            } else {
                printf(".");
            }
        }

        printf("\n");
    }
}


__device__ BBoard get_rook_premask(int idx) {

    BBoard result = 0;

    int x = (idx % 8);
    int y = (idx / 8);

    for (int i=-8; i<8; i++) {
        if (i == 0) continue;

        if (y + i > 0 && y + i < 7) {
            add_bit(result, x, y + i);
        }

        if (x + i > 0 && x + i < 7) {
            add_bit(result, x + i, y);
        } 
    }

    return result;
}

__device__ BBoard get_rook_attack_bits(BBoard indexed_mask, int x, int y) {

    bool slide_n = true;
    bool slide_s = true;
    bool slide_e = true;
    bool slide_w = true;

    BBoard result = 0;

    for (int i = 1; i < 8; i++) {

        if (slide_e && x + i < 8) {

            add_bit(result, x + i, y);

            if (has_bit(indexed_mask, x + i, y)) {
                slide_e = false;
            }
        }

        if (slide_w && x - i >= 0) {

            add_bit(result, x - i, y);

            if (has_bit(indexed_mask, x - i, y)) {
                slide_w = false;
            }
        }

        if (slide_n && y + 1 < 8) {

            add_bit(result, x, y + i);

            if (has_bit(indexed_mask, x, y + i)) {
                slide_n = false;
            }
        }

        if (slide_s && y - 1 >= 0) {

            add_bit(result, x, y - i);

            if (has_bit(indexed_mask, x, y - i)) {
                slide_s = false;
            }
        }

    }

    return result;
}

__device__ BBoard get_indexed_mask(BBoard pre_mask, int mask_number) {

    BBoard result = 0ULL;

    while (pre_mask > 0) {
        BBoard last_bit = pre_mask & -pre_mask;

        bool is_present = mask_number % 2 == 1;

        if (is_present) {
            result |= last_bit;
        }
        mask_number >>= 1;
        pre_mask ^= last_bit;
    }

    return result;
};



__global__ void init_magic_search(BBoard *d_indexed_mask, BBoard *d_attack_bits, int idx, int index_bits) {

    BBoard pre_mask = get_rook_premask(idx);

    int max_test_index = 1 << __popcll(pre_mask);

    int step = 0;
    int x = idx % 8;
    int y = idx / 8;

    while (step < max_test_index) {

        BBoard indexed_mask_e = get_indexed_mask(pre_mask, step);
        d_indexed_mask[step] = indexed_mask_e;

        BBoard attack_bits_e = get_rook_attack_bits(indexed_mask_e, x, y);
        d_attack_bits[step] = attack_bits_e;

        step += 1;
    }
}


__device__ bool check_rook_magic(BBoard *d_indexed_mask, BBoard *d_attack_bits, u64 magic, int max_test_index, int index_bits) {
    
    int index_size = 1ull << index_bits;

    if (index_size > ELEM_COUNT) {
        return false;
    }
    
    BBoard solution[ELEM_COUNT] {};

    int step = 0;
    int shift_bits = 64 - index_bits;

    while (step < max_test_index) {

        BBoard indexed_mask_e = d_indexed_mask[step];
        BBoard rook_attack_mask_e = d_attack_bits[step];

        int index = (indexed_mask_e * magic) >> shift_bits;

        BBoard cur_value = solution[index];

        if (cur_value > 0) {

            if (cur_value == rook_attack_mask_e) {
                step += 1;
                continue;
            } else {

                return false;
            }
        }

        solution[index] = rook_attack_mask_e;

        step += 1;
    }

    return true;
}


__global__ void find_magic(BBoard *d_indexed_mask, BBoard *d_attack_bits, int idx, int index_bits, u64 initial_counter, u64 *result) {

    int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

    BBoard pre_mask = get_rook_premask(idx);

    int max_test_index = 1ull << __popcll(pre_mask);

    if (*result != 0) {
        
        return;
    }

//////////////////////////////
    u64 magic_candidate = initial_counter + tid;

    bool is_good_magic = check_rook_magic(d_indexed_mask, d_attack_bits, magic_candidate, max_test_index, index_bits);

    if (is_good_magic) {
        printf("blockIdx = %d blockDim = %d threadIdx = %d tid * 2 = %d magic=0x%llx\n", blockIdx.x, blockDim.x, threadIdx.x, tid, magic_candidate);

        *result = magic_candidate;
    }


}


u64 getRandom()
{
    return (((u64)(unsigned int)rand() << 32) + (u64)(unsigned int)rand());
}

int main()
{
    /////////////////////////////////////////
    int idx = IDX;
    int index_bits = INDEX_BITS;
    u64 initial_counter = INITIAL_COUNTER;
    
    /////////////////////////////////////////

    BBoard *d_attack_bits;
    BBoard *d_indexed_mask;

    hipMalloc(&d_attack_bits, 4096 * sizeof(BBoard));
    hipMalloc(&d_indexed_mask, 4096 * sizeof(BBoard));

    init_magic_search<<<1, 1>>>(d_indexed_mask, d_attack_bits, idx, index_bits);

    printf("Start magic search\n");

    u64 result = 1;
    u64* d_result;

    int size = sizeof(u64);

    hipMalloc(&d_result, size);

    srand (time(NULL));
    u64 total_tests = 0;

    u64 initial_check_value = getRandom();

    initial_check_value = initial_counter;
    while (1) {

        u64 block_count = 10000;
        u64 block_size = 512;
        u64 sample_count = block_count * block_size;

        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

        find_magic<<<block_count, block_size>>>(d_indexed_mask, d_attack_bits, idx, index_bits, initial_check_value, d_result);

        hipMemcpy(&result, d_result, size, hipMemcpyDeviceToHost);

        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

        u64 speed = 1000000 * sample_count / std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
        total_tests += sample_count;

        printf("Testing: 0x%llx - 0x%llx, ", initial_check_value, initial_check_value + sample_count * 2);

        printf("%lld samples per second, %lld total tests\n", speed, total_tests);

        if (result != 0) {
            break;
        }


        initial_check_value += sample_count * 2;
    }

    hipFree(d_attack_bits);
    hipFree(d_indexed_mask);
    hipFree(d_result);

    printf("Magic search finished!\n");
    printf("Result: 0x%llx!\n", result);

}
